
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void transpose_matrix(int** mat, int **tran, const size_t rows, const size_t cols){

    size_t i = blockIdx.x; 
    size_t j = threadIdx.x;
    if (i < rows && j < cols)
	tran[j][i] = mat[i][j]; 
}

void fill_mat(int **mat, const size_t rows, const size_t cols);
void print_mat(int **mat, const size_t rows, const size_t cols);

#define N 5
#define M 6

int main(){
    
    int **mat = (int **) calloc(N, sizeof(int *));
    int **transp = (int **) calloc(M, sizeof(int *));
    size_t i;

    int **dev_mat, **dev_transp;
    hipMalloc((void***) &dev_mat, N*sizeof(int *));
    for (i = 0; i < N; ++i)
	hipMalloc((void**) &(mat[i]), M*sizeof(int));
    hipMemcpy(&dev_mat, &mat, N, hipMemcpyHostToDevice);

    hipMalloc((void***) &dev_transp, M*sizeof(int *));
    for (i = 0; i < M; ++i)
	hipMalloc((void**) &(transp[i]), N*sizeof(int));
    hipMemcpy(&dev_transp, &transp, M, hipMemcpyHostToDevice);

    for (i = 0; i < N; ++i)
	mat[i] = (int *) calloc(M, sizeof(int));

    for (i = 0; i < M; ++i)
	transp[i] = (int *) calloc(N, sizeof(int));

    fill_mat(mat, N, M);

    for (i = 0; i < N; ++i)
	hipMemcpy(&(dev_mat[i]), &(mat[i]), M, hipMemcpyHostToDevice);


    transpose_matrix<<<N,M>>>(dev_mat, dev_transp, N, M);

    for (i = 0; i < M; ++i)
	hipMemcpy(&(transp[i]), &(dev_transp[i]), N, hipMemcpyDeviceToHost);

    printf("Matrix:\n");
    print_mat(mat, N, M);
    printf("Transpose:\n");
    print_mat(transp, M, N);
    
}

void fill_mat(int **mat, const size_t rows, const size_t cols){

    size_t i;
    size_t j;
    for (i = 0; i < rows; ++i)
       for (j = 0; j < cols; ++j)
	    mat[i][j] = rand() % 100;	   
}

void print_mat(int **mat, const size_t rows, const size_t cols){

    size_t i, j;
    for (i = 0; i < rows; ++i){

       for (j = 0; j < cols; ++j)
	   printf("%d\t", mat[i][j]); 

       putchar('\n');
    }
}
