
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

# define N 128
# define N_THRDS 16

__global__ void init_mat (size_t *A_array, size_t **A, const size_t cols){

    size_t i = threadIdx.x;
    A[i] = (A_array + i*cols);
}

__global__ void transpose (size_t **A, size_t **B, const size_t n){

    size_t x = threadIdx.x;
    size_t y = blockIdx.x; 
    while (x < n){
    
	B[x][y] = A[y][x];
	x += blockDim.x;
    }
}

__global__ void fast_transpose(size_t **A, size_t **B, const size_t dim){

    __shared__ size_t a_block[N_THRDS];
    __shared__ size_t b_block[N_THRDS];

    if (threadIdx.x == 0 && threadIdx.y == 0){
    
    size_t i;
    for (i = 0; i < N_THRDS; ++i)
        a_block[i] = A[dim*blockIdx.y + i/dim][dim*blockIdx.x + i%dim];
    }
    __syncthreads();

    b_block[dim*threadIdx.y + threadIdx.x] = a_block[dim*threadIdx.x + threadIdx.y];

    __syncthreads();

    if(threadIdx.x == 0 && threadIdx.y == 0){
    
    size_t i;
    for (i = 0; i < N_THRDS; ++i)
        B[dim*blockIdx.x + i/dim][dim*blockIdx.y + i%dim] = b_block[i];
    }
}

void fill_mat(size_t *mat, const size_t rows, const size_t cols);
int is_transpose(size_t *mat, size_t *transp, const size_t n);
void print_is_transpose(size_t *mat, size_t *transp, const size_t n);

int main() {

  size_t* mat_array = (size_t*) malloc(N*N*sizeof(size_t));
  size_t* transp_array = (size_t*) malloc(N*N*sizeof(size_t));

  fill_mat(mat_array, N, N);

  size_t *dev_mat_array, *dev_transp_array;
  size_t **dev_mat, **dev_transp;

  hipMalloc( (void**)&dev_mat_array, N*N*sizeof(size_t) );
  hipMalloc( (void**)&dev_transp_array, N*N*sizeof(size_t) );
  hipMalloc( (void***)&dev_mat, N*sizeof(size_t) );
  hipMalloc( (void***)&dev_transp, N*sizeof(size_t) );

  hipMemcpy( dev_mat_array, mat_array, N*N*sizeof(size_t), hipMemcpyHostToDevice ); 

  init_mat<<< 1, N >>>(dev_mat_array, dev_mat,N);
  init_mat<<< 1, N >>>(dev_transp_array, dev_transp,N);

  transpose<<<N, N>>>(dev_mat, dev_transp, N);

  hipMemcpy( transp_array, dev_transp_array, N*N*sizeof(size_t),   hipMemcpyDeviceToHost );

  printf("Transpose result is: %d\n", is_transpose(mat_array, transp_array, N));

 size_t dim= (size_t)sqrt(N_THRDS);
  dim3 grid,block;
  grid.x=N/dim;
  grid.y=N/dim;
  block.x=dim;
  block.y=dim;

  fast_transpose<<< grid, block >>>(dev_mat, dev_transp,dim);

  hipMemcpy( transp_array, dev_transp_array, N*N*sizeof(size_t),   hipMemcpyDeviceToHost );

  printf("Fast transpose result is: %d\n", is_transpose(mat_array, transp_array, N));

//  printf("Matrix:\n");
//  size_t i;
//  for(i=0;i<N*N;i++){
//    if(i%N==0 && i!=0)printf("\n");
//      printf("%d ", mat_array[i]);
//  }
//  printf("\n");
//
//  printf("Transpose:\n");
//  for(i=0;i<N*N;i++){
//    if(i%N==0 && i!=0)printf("\n");
//
//printf("%d ", transp_array[i]);
//  }
//  printf("\n");
//

    
 // print_is_transpose(mat_array, transp_array, N);

  free(mat_array); free(transp_array);
  hipFree( dev_mat_array ); hipFree( dev_transp_array ); hipFree(dev_mat);hipFree(dev_transp);
  return 0;
}
void fill_mat(size_t *mat, const size_t rows, const size_t cols){

    size_t i;
    for (i = 0; i < rows*cols; ++i)
        mat[i] = rand() % 100;	   
}
int is_transpose(size_t *mat, size_t *transp, const size_t n){
    
    size_t i, j;
    for (i = 0; i < n; ++i)
    for (j = 0; j < n; ++j)
        if (mat[i*n + j] != transp[j*n + i])
	return 0;
    return 1;
}
void print_is_transpose(size_t *mat, size_t *transp, const size_t n){
    
    size_t i, j;
    for (i = 0; i < n; ++i){
    for (j = 0; j < n; ++j)
        printf("%d",(mat[i*n + j] != transp[j*n + i]) ? 0 : 1);
    putchar('\n');
    }
}
